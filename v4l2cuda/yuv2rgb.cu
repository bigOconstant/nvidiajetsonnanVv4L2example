#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>
#include "yuv2rgb.cuh"

__device__ inline float clamp(float val, float mn, float mx)
{
	return (val >= mn)? ((val <= mx)? val : mx) : mn;
}

__global__ void gpuConvertYUYVtoRGB_kernel(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	for (int i = 0; i < height; ++i) {
		int y0 = src[i*width*2+idx*4+0];
		int cb = src[i*width*2+idx*4+1];
		int y1 = src[i*width*2+idx*4+2];
		int cr = src[i*width*2+idx*4+3];

		dst[i*width*3+idx*6+0] = clamp(1.164f * (y0 - 16) + 1.596f * (cr - 128)                      , 0.0f, 255.0f);
		dst[i*width*3+idx*6+1] = clamp(1.164f * (y0 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), 0.0f, 255.0f);
		dst[i*width*3+idx*6+2] = clamp(1.164f * (y0 - 16)                       + 2.018f * (cb - 128), 0.0f, 255.0f);

		dst[i*width*3+idx*6+3] = clamp(1.164f * (y1 - 16) + 1.596f * (cr - 128)                      , 0.0f, 255.0f);
		dst[i*width*3+idx*6+4] = clamp(1.164f * (y1 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), 0.0f, 255.0f);
		dst[i*width*3+idx*6+5] = clamp(1.164f * (y1 - 16)                       + 2.018f * (cb - 128), 0.0f, 255.0f);
	}
}

void gpuConvertYUYVtoRGB(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	unsigned char *d_src = NULL;
	unsigned char *d_dst = NULL;
	size_t planeSize = width * height * sizeof(unsigned char);

	unsigned int flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);
	bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped) {
		d_src = src;
		hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_src, planeSize * 2);
		hipMemcpy(d_src, src, planeSize * 2, hipMemcpyHostToDevice);
	}
	if (dstIsMapped) {
		d_dst = dst;
		hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_dst, planeSize * 3);
	}

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	gpuConvertYUYVtoRGB_kernel<<<numBlocks, blockSize>>>(d_src, d_dst, width, height);
	hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachHost);
	hipStreamSynchronize(NULL);

	if (!srcIsMapped) {
		hipMemcpy(dst, d_dst, planeSize * 3, hipMemcpyDeviceToHost);
		hipFree(d_src);
	}
	if (!dstIsMapped) {
		hipFree(d_dst);
	}
}
